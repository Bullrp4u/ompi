#include "ompi_config.h"
#include "coll_adapt_cuda.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdarg.h>

static int coll_adapt_cuda_kernel_enabled = 0;

int coll_adapt_cuda_init(void)
{
    int device;
    hipError_t cuda_err;

    cuda_err = hipGetDevice(&device);
    if( hipSuccess != cuda_err ) {
       // OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Cannot retrieve the device being used. Drop CUDA support!\n"));
        return -1;
    }
    
    coll_adapt_cuda_kernel_enabled = 1;
    return 0;
}

int coll_adapt_cuda_fini(void)
{
    coll_adapt_cuda_kernel_enabled = 0;
    return 0;
}

int coll_adapt_cuda_is_gpu_buffer(const void *ptr)
{
    hipMemoryType memType;
    hipDeviceptr_t dbuf = (hipDeviceptr_t)ptr;
    int res;

    res = hipPointerGetAttribute(&memType, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dbuf);
    if (res != hipSuccess) {
        /* If we cannot determine it is device pointer,
         * just assume it is not. */
      //  OPAL_OUTPUT_VERBOSE((1, opal_datatype_cuda_output, "!!!!!!! %p is not a gpu buffer. Take no-CUDA path!\n", ptr));
        return 0;
    }
    /* Anything but hipMemoryTypeDevice is not a GPU memory */
    return (memType == hipMemoryTypeDevice) ? 1 : 0;
}